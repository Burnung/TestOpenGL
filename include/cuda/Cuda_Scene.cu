#include "hip/hip_runtime.h"
#include"Cuda_Scene.cuh"
#include <hip/hip_runtime_api.h>
#include <helper_string.h>
#include "../model.h"
#include"../PHO_ViewPort.h"

__shared__ int* dev_TriIndex[1 << MAX_CUDA_KDTRE_DEPTH];
__shared__ CUDA_KDTree* m_devTree[100];
__shared__ Cuda_Sphere* m_devSphere[100];

PHO_DEFINE_SINGLETON_NO_CTOR(Cuda_Scene);


__global__ void SetDevTree(CUDA_KDTree *dev_Tree, int**TriIndex){
	unsigned int index = blockIdx.x*blockDim.x + threadIdx.x;

	if (index >= (1 << MAX_CUDA_KDTRE_DEPTH))
		return;
	if (dev_Tree->m_TreeNode[index].isLeaf == 1){
		dev_Tree->m_TreeNode[index].m_TriIndex = TriIndex[index];
		dev_Tree->m_TreeNode[index].m_TriList = dev_Tree->m_TriList;
		printf("%d %d\n", index, dev_Tree->m_TreeNode[index].m_Num);
	}
}

__host__ void ReleaseHostTree(CUDA_KDTree *kdTree){
	CUDA_TreeNode *tmpNode = kdTree->m_TreeNode;
	if (tmpNode){
		for (int i = 0; i < 1 << MAX_CUDA_KDTRE_DEPTH; i++){
			if (tmpNode[i].isLeaf == 1 && tmpNode[i].m_TriIndex)
				delete[] tmpNode[i].m_TriIndex;
		}
		delete[] tmpNode;
	}
	if (kdTree->m_TriList)
		delete[] kdTree->m_TriList;
	delete kdTree;
}

__host__ void BuilKdTree(CUDA_KDTree *m_KDTree, CUDA_Triangle* Triangles, int TriNum){
	bool *IsInLeaf = new bool[TriNum];
	memset(IsInLeaf, 0, sizeof(bool)* TriNum);
	m_KDTree->m_TreeNode = new CUDA_TreeNode[1 << MAX_CUDA_KDTRE_DEPTH];

	m_KDTree->m_TriNum = TriNum;
	m_KDTree->m_TriList = Triangles;
	//��ʼ����kdtree
	 //�õ�ÿ�������εİ�Χ��
	std::vector<CUDA_AABB> m_AABB(TriNum);
	for (int i = 0; i < TriNum; i++)
		m_AABB[i] = GetAABBFromTri(Triangles[i]);
	//�����ܵİ�Χ��
	CUDA_AABB AllBound = m_AABB[0];
	for (int i = 1; i < TriNum; i++)
		ExpandBox(AllBound, m_AABB[i]);
	//����ȫ����
	std::vector<int> triIndx(TriNum);
	for (int i = 0; i < TriNum; i++)
		triIndx[i] = i;

	//�ݹ鹹��
	buildKdNode(m_KDTree->m_TreeNode, Triangles, m_AABB, AllBound, triIndx, 1, 0);

}
__host__ void buildKdNode(CUDA_TreeNode* kdNode, CUDA_Triangle* Triangles, std::vector<CUDA_AABB>& allAABB, CUDA_AABB &ALLBound, std::vector<int>&TriIndex, int depth, int NodeIndex){
	kdNode[NodeIndex].m_AABB = ALLBound;
	//����������Ҷ�ӽڵ�
	if (depth >= MAX_CUDA_KDTRE_DEPTH || TriIndex.size() <= MIN_CUDA_KDTRE_COUNT){
		kdNode[NodeIndex].isLeaf = true;
		kdNode[NodeIndex].m_Num = TriIndex.size();
		kdNode[NodeIndex].m_TriIndex = new int[TriIndex.size()];
		memcpy(kdNode[NodeIndex].m_TriIndex, &TriIndex[0], sizeof(int)*TriIndex.size());
		kdNode[NodeIndex].m_TriList = Triangles;
		return;
	}
	float MidPos;
	int Axi = GetMaxAxi(ALLBound, MidPos);
	std::vector<int> leftIndex(0);
	std::vector<int> rightIndex(0);
	switch (Axi){
	case X_AXIS:
		for (int i = 0; i < TriIndex.size(); i++)
			Triangles[TriIndex[i]].m_MidPoint.x < MidPos ? leftIndex.push_back(TriIndex[i]) : rightIndex.push_back(TriIndex[i]);
		break;
	case Y_AXIS:
		for (int i = 0; i < TriIndex.size(); i++)
			Triangles[TriIndex[i]].m_MidPoint.y < MidPos ? leftIndex.push_back(TriIndex[i]) : rightIndex.push_back(TriIndex[i]);
		break;
	case Z_AXIS:
		for (int i = 0; i < TriIndex.size(); i++)
			Triangles[TriIndex[i]].m_MidPoint.z < MidPos ? leftIndex.push_back(TriIndex[i]) : rightIndex.push_back(TriIndex[i]);
		break;
	}
	if (leftIndex.size() == TriIndex.size() || rightIndex.size() == TriIndex.size()){
		kdNode[NodeIndex].isLeaf = true;
		kdNode[NodeIndex].m_Num = TriIndex.size();
		kdNode[NodeIndex].m_TriIndex = new int[TriIndex.size()];
		memcpy(kdNode[NodeIndex].m_TriIndex, &TriIndex[0], sizeof(int)*TriIndex.size());
		kdNode[NodeIndex].m_TriList = Triangles;
		return;
	}
	CUDA_AABB leftAABB = allAABB[leftIndex[0]];
	for (int i = 1; i < leftIndex.size(); i++)
		ExpandBox(leftAABB, allAABB[leftIndex[i]]);

	CUDA_AABB rightAABB = allAABB[rightIndex[0]];
	for (int i = 1; i < rightIndex.size(); i++)
		ExpandBox(rightAABB,allAABB[rightIndex[i]]);

	kdNode[NodeIndex].m_LeftIndex = 2 * NodeIndex + 1;
	buildKdNode(kdNode, Triangles, allAABB, leftAABB, leftIndex, depth + 1, kdNode[NodeIndex].m_LeftIndex);

	kdNode[NodeIndex].m_RightIndex = 2 * NodeIndex + 2;
	buildKdNode(kdNode, Triangles, allAABB, rightAABB, rightIndex, depth + 1, kdNode[NodeIndex].m_RightIndex);

}

Cuda_Scene::Cuda_Scene(){
	m_SphereNum = 0;
	m_KdTreeNum = 0;
	m_MatNum = 0;
	m_Dev_Spheres.resize(0);
	m_Dev_KdTree.resize(0);
	m_Host_Tracer = NULL;
	m_Dev_Tracer = NULL;
	m_Dev_Mat = NULL;
	AllIsOk = false;
}

Cuda_Scene::~Cuda_Scene(){
	ReleaseWorld();
}

void Cuda_Scene::SetCudaSceneMat(std::vector<GL_Material*>&mats){
	m_MatNum = mats.size();
	Cuda_Material *TmpMat;
	TmpMat = new Cuda_Material[m_MatNum];
	for (int i = 0; i < m_MatNum; i++)
		TmpMat[i] = GetCudaMatFromMat(mats[i]);
	checkCudaErrors(hipMalloc((void **)&(m_Dev_Mat), mats.size() * sizeof(Cuda_Material)));
	checkCudaErrors(hipMemcpy(m_Dev_Mat, TmpMat, mats.size()&sizeof(Cuda_Material), hipMemcpyHostToDevice));
	delete[] TmpMat;

}
void Cuda_Scene::AddSphere(SphereObj *Sph){
	//���� cuda_spherer
	Cuda_Sphere* tmpSphere = new Cuda_Sphere;// GetSphereFromObj(Sph);

	//���俽����GPU
	Cuda_Sphere *tmp_dev_Sph;
	checkCudaErrors(hipMalloc((void**)&tmp_dev_Sph, sizeof(Cuda_Sphere)));
	checkCudaErrors(hipMemcpy(tmp_dev_Sph, tmpSphere, sizeof(Cuda_Sphere), hipMemcpyHostToDevice));

	m_Dev_Spheres.push_back(tmp_dev_Sph);

	delete tmpSphere;
}

void Cuda_Scene::AddKdTree(std::vector<Triangle*>& tris){
	CUDA_Triangle *cudaTris = new CUDA_Triangle[tris.size()];
	for (int i = 0; i < tris.size(); i++){
		GetCudaTrifromTri(cudaTris[i], tris[i]);
	}

	CUDA_KDTree *hostTree = new CUDA_KDTree;
	//��cpu����kdtree
	BuilKdTree(hostTree, cudaTris, tris.size());

	//��gup�����ڴ�
	CUDA_Triangle *dev_cudaTris;
	CUDA_KDTree *dev_cudaTree;
	checkCudaErrors(hipMalloc((void**)&dev_cudaTris, sizeof(CUDA_Triangle)*tris.size()));
	checkCudaErrors(hipMemcpy(dev_cudaTris, cudaTris, sizeof(CUDA_Triangle)*tris.size(), hipMemcpyHostToDevice));

	CUDA_TreeNode *dev_treeNode;
	checkCudaErrors(hipMalloc((void**)&dev_treeNode, sizeof(CUDA_TreeNode)*(1 << MAX_CUDA_KDTRE_DEPTH)));
	checkCudaErrors(hipMemcpy(dev_treeNode, hostTree->m_TreeNode, sizeof(CUDA_TreeNode)*(1 << MAX_CUDA_KDTRE_DEPTH), hipMemcpyHostToDevice));

	//kdtree
	CUDA_KDTree *dev_tree;
	checkCudaErrors(hipMalloc((void**)&dev_tree, sizeof(CUDA_KDTree)));
	hostTree->m_TriList = dev_cudaTris;
	CUDA_TreeNode *tmpTreeNode = hostTree->m_TreeNode;
	hostTree->m_TreeNode = dev_treeNode;
	checkCudaErrors(hipMemcpy(dev_tree, hostTree, sizeof(CUDA_KDTree), hipMemcpyHostToDevice));
	hostTree->m_TriList = cudaTris;
	hostTree->m_TreeNode = tmpTreeNode;

	//����������
	std::vector<int*> tmpindex;
	for (int i = 0; i < (1 << MAX_CUDA_KDTRE_DEPTH); i++){
		dev_TriIndex[i] = NULL;
		if (tmpTreeNode[i].isLeaf == 1){
			checkCudaErrors(hipMalloc((void**)&dev_TriIndex[i], sizeof(int)*tmpTreeNode[i].m_Num));
			checkCudaErrors(hipMemcpy(dev_TriIndex[i], tmpTreeNode[i].m_TriIndex, sizeof(int)*tmpTreeNode[i].m_Num, hipMemcpyHostToDevice));
			tmpindex.push_back(dev_TriIndex[i]);
		}
	}
	m_allTriIndex.push_back(tmpindex);

	dim3 dimBlock(32, 1, 1);
	dim3 dimGrid((1 << (MAX_CUDA_KDTRE_DEPTH)) / dimBlock.x + 1, 1, 1);

	int y = 1024 / 8;

	SetDevTree << < 8, y, 1 >> >(dev_tree, dev_TriIndex);

	ReleaseHostTree(hostTree);

	m_DevTrisList.push_back(dev_cudaTris);
	m_Dev_KdTree.push_back(dev_tree);
}

void Cuda_Scene::ReleaseWorld(){
	for (auto item : m_Dev_Spheres)
		hipFree(item);
	for (auto item : m_allTriIndex){
		for (auto litem : item)
			hipFree(litem);
		item.clear();
	}
	for (auto item : m_Dev_KdTree){
		hipFree(item);
	}
	for (auto item : m_DevTrisList)
		hipFree(item);

	delete(m_Host_Tracer);
	hipFree(m_Dev_Mat);
	hipFree(m_Dev_Tracer);


	m_Dev_Spheres.clear();
	m_DevTrisList.clear();
	m_Dev_Spheres.clear();
	m_allTriIndex.clear();
}
void Cuda_Scene::SetTracer(PHO_ViewPort* tmpView){

	//Ϊdev�����ڴ�
	checkCudaErrors(hipMalloc((void**)&m_Dev_Tracer, sizeof(Cuda_TracerSet)));

	m_Host_Tracer->m_WindowWidth = tmpView->GetWidth();
	m_Host_Tracer->m_WindowHeight = tmpView->GetHeight();

	//�������ɹ���ʱ��һЩ����
	m_Host_Tracer->m_Width_recp = 1.0f / (m_Host_Tracer->m_WindowWidth *1.0f);
	m_Host_Tracer->m_Height_recp = 1.0f / (m_Host_Tracer->m_WindowHeight *1.0f);
	m_Host_Tracer->m_Ratio = (m_Host_Tracer->m_WindowWidth *1.0f) / (m_Host_Tracer->m_WindowHeight *1.0f);

	m_Host_Tracer->m_FovS = 1.0 / tanf(tmpView->GetFovy() * 0.5);
	m_Host_Tracer->m_X_Spacing = m_Host_Tracer->m_Width_recp * (m_Host_Tracer->m_Ratio) * (m_Host_Tracer->m_FovS);
	m_Host_Tracer->m_Y_Spacing = m_Host_Tracer->m_Height_recp * (m_Host_Tracer->m_FovS);
	m_Host_Tracer->m_X_Spacing_Half = m_Host_Tracer->m_X_Spacing * 0.5f;
	m_Host_Tracer->m_Y_Spacing_Half = m_Host_Tracer->m_Y_Spacing * 0.5f;
	UpDateTracer(tmpView);

}
void Cuda_Scene::UpDateTracer(PHO_ViewPort* tmpView){
	m_Host_Tracer->m_CamPos = tmpView->GetCameraPos();
	m_Host_Tracer->m_CamTarVec = glm::normalize(tmpView->GetCameraLookVec());
	m_Host_Tracer->m_CamYVec = glm::normalize(tmpView->GetCameraUpVec());
	//����ˮƽ�ʹ�ֱ���򡣡� x,y,z ���μ���
	m_Host_Tracer->m_CamXVec = glm::cross(m_Host_Tracer->m_CamYVec, m_Host_Tracer->m_CamTarVec);
	m_Host_Tracer->m_CamXVec = glm::normalize(m_Host_Tracer->m_CamXVec);

	m_Host_Tracer->m_CamYVec = glm::normalize(glm::cross(m_Host_Tracer->m_CamTarVec, m_Host_Tracer->m_CamXVec));

	checkCudaErrors(hipMemcpy(m_Dev_Tracer, m_Host_Tracer, sizeof(Cuda_TracerSet), hipMemcpyHostToDevice));


}


__global__ void TraceAll(Cuda_TracerSet *m_Tracer, int samples){

	int nSamples = blockDim.x * blockIdx.y + blockIdx.x;
	if (nSamples >= samples)
		return;
	int x = threadIdx.x;
	int y = threadIdx.y;
	CUDA_Ray ray;


	
}


void Cuda_Scene::GoTrace(int samples){
	if (!AllIsOk){
		for (int i = 0; i < m_Dev_Spheres.size(); i++)
			m_devSphere[i] = m_Dev_Spheres[i];
		for (int i = 0; i < m_Dev_KdTree.size(); i++)
			m_devTree[i] = m_Dev_KdTree[i];
		AllIsOk = true;
	}

	dim3 dimBlock(4,samples/4+1 , 1);
	dim3 dimGrid(m_Host_Tracer->m_WindowWidth, m_Host_Tracer->m_WindowHeight);





}