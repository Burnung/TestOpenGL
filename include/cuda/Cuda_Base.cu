﻿#include "cuda_Base.cuh"
#include <hip/hip_runtime_api.h>

#include "../model.h"
#include "../GL_Scene.h"

__host__ CUDA_AABB GetAABBFromTri(CUDA_Triangle& tri){

	glm::vec3 MinPos;
	glm::vec3 MaxPos;
	glm::vec3 p1 = tri.m_p1.pos;
	glm::vec3 p2 = tri.m_p2.pos;
	glm::vec3 p3 = tri.m_p3.pos;

	MinPos.x = (p1.x < p2.x) && (p1.x < p3.x) ? p1.x : p2.x < p3.x ? p2.x : p3.x;
	MinPos.y = (p1.y < p2.y) && (p1.y < p3.y) ? p1.y : p2.y < p3.y ? p2.y : p3.y;
	MinPos.z = (p1.z < p2.z) && (p1.z < p3.z) ? p1.z : p2.z < p3.z ? p2.z : p3.z;

	MaxPos.x = (p1.x > p2.x) && (p1.x > p3.x) ? p1.x : p2.x > p3.x ? p2.x : p3.x;
	MaxPos.y = (p1.y > p2.y) && (p1.y > p3.y) ? p1.y : p2.y > p3.y ? p2.y : p3.y;
	MaxPos.z = (p1.z > p2.z) && (p1.z > p3.z) ? p1.z : p2.z > p3.z ? p2.z : p3.z;
	return CUDA_AABB(MinPos, MaxPos);
}
__host__ void ExpandBox(CUDA_AABB &ret, CUDA_AABB &tmp){
	ret.m_MinPos.x = ret.m_MinPos.x < tmp.m_MinPos.x ? ret.m_MinPos.x : tmp.m_MinPos.x;
	ret.m_MinPos.y = ret.m_MinPos.y < tmp.m_MinPos.y ? ret.m_MinPos.y : tmp.m_MinPos.y;
	ret.m_MinPos.z = ret.m_MinPos.z < tmp.m_MinPos.z ? ret.m_MinPos.z : tmp.m_MinPos.z;

	ret.m_MaxPos.x = ret.m_MaxPos.x > tmp.m_MaxPos.x ? ret.m_MaxPos.x : tmp.m_MaxPos.x;
	ret.m_MaxPos.y = ret.m_MaxPos.y > tmp.m_MaxPos.y ? ret.m_MaxPos.y : tmp.m_MaxPos.y;
	ret.m_MaxPos.z = ret.m_MaxPos.z > tmp.m_MaxPos.z ? ret.m_MaxPos.z : tmp.m_MaxPos.z;
}

__host__ int GetMaxAxi(CUDA_AABB &aabb, float &MidPos){
	glm::vec3 Diff = aabb.m_MaxPos - aabb.m_MinPos;
	glm::vec3 MidVPos = (aabb.m_MaxPos + aabb.m_MinPos) * 0.5f;
	if (Diff.x > Diff.y && Diff.x > Diff.z){
		MidPos = MidVPos.x;
		return X_AXIS;
	}
	else if (Diff.y > Diff.z){
		MidPos = MidVPos.y;
		return Y_AXIS;
	}
	else{
		MidPos = MidVPos.z;
		return Z_AXIS;
	}
	return X_AXIS;
}

__host__ Cuda_Material GetCudaMatFromMat(GL_Material*mat){
	Cuda_Material ret;
	ret.m_RenderType = mat->m_RenderType;
	ret.m_MaterialType = mat->m_MaterialType;

	ret.m_Refra = mat->m_Refra; //������  ���ڵ���1
	ret.m_colour = mat->m_colour;
	ret.m_emission = mat->m_emission;
	return ret;
}
__host__ Cuda_Sphere* GetSphereFromObj(SphereObj* Obj){

	Cuda_Sphere* ret = new Cuda_Sphere(Obj->getPos(), Obj->m_Raduis);
	ret->m_MatIndex = GL_Scene::Instance().GetMatIndex(Obj->GetMat());

	return ret;
}

__host__ void GetCudaTrifromTri(CUDA_Triangle &cuda_tri, Triangle* tri){
	cuda_tri.m_p1 = tri->m_p1;
	cuda_tri.m_p2 = tri->m_p2;
	cuda_tri.m_p3 = tri->m_p3;
	cuda_tri.m_Normal = tri->m_Normal;
	cuda_tri.m_MidPoint = ComputTriMidPoint(cuda_tri);
	cuda_tri.m_MatIndex = GL_Scene::Instance().GetMatIndex(tri->m_PMaterial);
}
__host__ glm::vec3 ComputTriMidPoint(CUDA_Triangle& tri){
	glm::vec3 ret;
	for (int i = 0; i < 3; i++)
		ret[i] = (tri.m_p1.pos[i] + +tri.m_p2.pos[i] + tri.m_p2.pos[i]);
	return ret;
}

__host__ void CUDA_InitCuda(){

	int device_count = 0;
	int device = -1;
	checkCudaErrors(hipGetDeviceCount(&device_count));

	for (int i = 0; i < device_count; ++i)
	{
		hipDeviceProp_t properties;
		checkCudaErrors(hipGetDeviceProperties(&properties, i));

		if (properties.major > 3 || (properties.major == 3 && properties.minor >= 5))
		{
			device = i;
			std::cout << "Running on GPU " << i << " (" << properties.name << ")" << std::endl;
			break;
		}

		std::cout << "GPU " << i << " (" << properties.name << ") does not support CUDA Dynamic Parallelism" << std::endl;
	}
	if (device == -1){
		std::cerr << "cdpSimpleQuicksort requires GPU devices with compute SM 3.5 or higher.  Exiting..." << std::endl;
		exit(EXIT_WAIVED);
	}
	hipSetDevice(device);

}